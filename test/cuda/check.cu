#include "hip/hip_runtime.h"
/**
 *  @file   check.c
 *  @author Kai Keller (kellekai@gmx.de)
 *  @date   June, 2017
 *  @brief  FTI testing program.
 *
 *	The program may test the correct behaviour for checkpoint
 *	and restart for all configurations. The recovered data is also
 *	tested upon correct data fields.
 *
 *	The program takes four arguments:
 *	  - arg1: FTI configuration file
 *	  - arg2: Interrupt yes/no (1/0)
 *	  - arg3: Checkpoint level (1, 2, 3, 4)
 *	  - arg4: different ckpt. sizes yes/no (1/0)
 *
 * If arg2 = 0, the program simulates a clean run of FTI:
 *    FTI_Init
 *    FTI_Protect
 *    if FTI_Status = 0
 *      FTI_Checkpoint
 *    else
 *      FTI_Recover
 *    FTI_Finalize
 *
 * If arg2 = 1, the program simulates an execution failure:
 *    FTI_Init
 *    FTI_Protect
 *    if FTI_Status = 0
 *      exit(10)
 *    else
 *      FTI_Recover
 *    FTI_Finalize
 *
 */

#include "mpi.h"
#include "fti.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <unistd.h>

#define CUDA_CALL_SAFE(f)                                                                       \
do {                                                                                            \
    hipError_t _e = f;                                                                          \
    if(_e != hipSuccess) {                                                                    \
      fprintf(stderr, "Cuda error %d %s:: %s\n", __LINE__, __func__, hipGetErrorString(_e));  \
      exit(EXIT_FAILURE);                                                                       \
    }                                                                                           \
} while(0)

#define BLOCK_SIZE 1024

//#define N 100000
//#define N ((size_t)1 << 25)
#define N ((size_t)1 << 20)
#define CNTRLD_EXIT 10
#define RECOVERY_FAILED 20
#define DATA_CORRUPT 30
#define KEEP 2
#define RESTART 1
#define INIT 0

/**
 * function prototypes
 */

/*-------------------------------------------------------------------------*/
/**
    @brief      Initialize test data
    @param      [out] A				Unit vector (1, 1, ....., 1)
    @param      [out] B				Random vector
    @param      [in] asize			Dimension

	Initializes A with 1's and B with random numbers r,  0 <= r <= 5.
	Dimension of both vectors is 'asize'
 **/
/*-------------------------------------------------------------------------*/
void init_arrays(double* A, double* B, size_t asize);

/*-------------------------------------------------------------------------*/
/**
    @brief      [CUDA] Multiplies components of A and B and stores result into A
    @param      [in/out] A			Unit vector (1, 1, ....., 1)
    @param      [in] B				Random vector
    @param      [in] asize			Dimension

    After function call, A equals B.
 **/
/*-------------------------------------------------------------------------*/
__global__
void vecmult(double* A, double* B, size_t asize);

/*-------------------------------------------------------------------------*/
/**
    @brief      Validifies the recovered data
    @param      [in] A			    A returned from vecmult
    @param      [in] B_chk			POSIX Backup of B
    @param      [in] asize			Dimension
    @return     integer             0 if successful, -1 else.

    Checks entry for entry if A equals the POSIX Backup of B, B_chk, from
    the preceding execution. This function must be called after the call to
    vecmult(A, B, asize).
 **/
/*-------------------------------------------------------------------------*/
int validify(double* A, double* B_chk, size_t asize);

/*-------------------------------------------------------------------------*/
/**
    @brief      Writes 'B' and 'asize' to file, using POSIX fwrite.
    @param      [in] B              Random array B from init_array call
    @param      [in] asize			Dimension
    @param      [in] rank           FTI application rank
 **/
/*-------------------------------------------------------------------------*/
int write_data(double* B, size_t* asize, int rank);

/*-------------------------------------------------------------------------*/
/**
    @brief      Recovers 'B' and 'asize' to 'B_chk' and 'asize_chk' from file,
                using POSIX fread.
    @param      [out] B_chk         B backup
    @param      [out] asize_chk     Dimension backup
    @param      [in] rank           FTI application rank
    @param      [in] asize			Dimension
    @return     integer             0 if successful, -1 else.

    Before recovering B, the function checks if 'asize_chk' equals 'asize',
    to prevent SIGSEGV. If not 'asize_chk' = 'asize' it returns -1.
 **/
/*-------------------------------------------------------------------------*/
int read_data(double* B_chk, size_t* asize_chk, int rank, size_t asize);

/**
 * main
 */

int main(int argc, char* argv[]) {

    unsigned char parity, crash, level, state, diff_sizes;
    int FTI_APP_RANK, result, tmp, success = 1;
    double *h_A, *h_B, *B_chk;
    double *d_A, *d_B;

    size_t asize, asize_chk;
    size_t asize_with_dt;

    int blocks_per_grid, threads_per_block;

    srand(time(NULL));

    MPI_Init(&argc, &argv);
    result = FTI_Init(argv[1], MPI_COMM_WORLD);
    if (result == FTI_NREC) {
        exit(RECOVERY_FAILED);
    }

    crash = atoi(argv[2]);
    level = atoi(argv[3]);
    diff_sizes = atoi(argv[4]);

    MPI_Comm_rank(FTI_COMM_WORLD,&FTI_APP_RANK);

    asize = N;

    if (diff_sizes) {
        parity = FTI_APP_RANK%7;

        switch (parity) {

            case 0:
                asize = N;
                break;

            case 1:
                asize = 2*N;
                break;

            case 2:
                asize = 3*N;
                break;

            case 3:
                asize = 4*N;
                break;

            case 4:
                asize = 5*N;
                break;

            case 5:
                asize = 6*N;
                break;

            case 6:
                asize = 7*N;
                break;

        }
    }

    asize_with_dt = asize * sizeof(double);

    threads_per_block = BLOCK_SIZE;
    blocks_per_grid = (asize + threads_per_block - 1) / threads_per_block;
    
    state = FTI_Status();

    if ( (FTI_APP_RANK == 0) && (state == INIT) ) { printf(
    "<< ALLOCATE GPU MEMORY >>\n"
    "## \n"
    "## h_A = (double*) malloc(asize_with_dt);\n"
    "## h_B = (double*) malloc(asize_with_dt);\n"
    "## \n"
    "## CUDA_CALL_SAFE(hipMalloc(&d_A, asize_with_dt));\n"
    "## CUDA_CALL_SAFE(hipMalloc(&d_B, asize_with_dt));\n"
    "## \n"
    "## FTI_Protect(0, d_A, asize, FTI_DBLE);\n"
    "## FTI_Protect(1, d_B, asize, FTI_DBLE);\n"
    "## FTI_Protect(2, &asize, 1, FTI_INTG);\n");}

    h_A = (double*) malloc(asize_with_dt);
    h_B = (double*) malloc(asize_with_dt);

    CUDA_CALL_SAFE(hipMalloc(&d_A, asize_with_dt));
    CUDA_CALL_SAFE(hipMalloc(&d_B, asize_with_dt));

    FTI_Protect(0, d_A, asize, FTI_DBLE);
    FTI_Protect(1, d_B, asize, FTI_DBLE);
    FTI_Protect(2, &asize, 1, FTI_INTG);

    if (state == INIT) {
        if (FTI_APP_RANK == 0) { printf(
        "<< SEND DATA TO GPU MEMORY PERFORM CHECKPOINT AND SIMULATE CRASH >>\n"
        "## \n"
        "## init_arrays(h_A, h_B, asize);\n"
        "## write_data(h_B, &asize, FTI_APP_RANK);\n"
        "## CUDA_CALL_SAFE(hipMemcpy(d_A, h_A, asize_with_dt, hipMemcpyHostToDevice));\n"
        "## CUDA_CALL_SAFE(hipMemcpy(d_B, h_B, asize_with_dt, hipMemcpyHostToDevice));\n"
        "## MPI_Barrier(FTI_COMM_WORLD);\n"
        "## FTI_Checkpoint(1,level);\n"
        "## if (crash && FTI_APP_RANK == 0) {\n"
        "##     exit(CNTRLD_EXIT);\n"
        "## }\n"); }
        init_arrays(h_A, h_B, asize);
        write_data(h_B, &asize, FTI_APP_RANK);
        CUDA_CALL_SAFE(hipMemcpy(d_A, h_A, asize_with_dt, hipMemcpyHostToDevice));
        CUDA_CALL_SAFE(hipMemcpy(d_B, h_B, asize_with_dt, hipMemcpyHostToDevice));
        MPI_Barrier(FTI_COMM_WORLD);
        FTI_Checkpoint(1,level);
        //sleep(5);
        if (crash && FTI_APP_RANK == 0) {
            MPI_Abort(MPI_COMM_WORLD,0);
        }
    }

    if ( state == RESTART || state == KEEP ) {
        result = FTI_Recover();
        if (result != FTI_SCES) {
            exit(RECOVERY_FAILED);
        }
        B_chk = (double*) malloc(asize_with_dt);
        result = read_data(B_chk, &asize_chk, FTI_APP_RANK, asize);
        MPI_Barrier(FTI_COMM_WORLD);
        if (result != 0) {
            exit(DATA_CORRUPT);
        }
    }

    /*
     * on INIT, B is initialized randomly
     * on RESTART or KEEP, B is recovered and must be equal to B_chk
     */

    if ( (FTI_APP_RANK == 0) ) { printf(
    "<< VALIDATE RESULT AFTER RECOVERY >>\n"
    "## \n"
    "## vecmult<<< blocks_per_grid, threads_per_block >>>(d_A, d_B, asize);\n"
    "## CUDA_CALL_SAFE(hipDeviceSynchronize());\n"
    "## CUDA_CALL_SAFE(hipMemcpy(h_A, d_A, asize_with_dt, hipMemcpyDeviceToHost));\n"
    "## result = validify(h_A, B_chk, asize);\n"
    "## result += (asize_chk == asize) ? 0 : -1;\n"
    "## MPI_Allreduce(&result, &result_glb, 1, MPI_INT, MPI_SUM, FTI_COMM_WORLD);\n"); }

    vecmult<<< blocks_per_grid, threads_per_block >>>(d_A, d_B, asize);
    CUDA_CALL_SAFE(hipDeviceSynchronize());
    CUDA_CALL_SAFE(hipMemcpy(h_A, d_A, asize_with_dt, hipMemcpyDeviceToHost));

    if (state == RESTART || state == KEEP) {
        result = validify(h_A, B_chk, asize);
        result += (asize_chk == asize) ? 0 : -1;
        MPI_Allreduce(&result, &tmp, 1, MPI_INT, MPI_SUM, FTI_COMM_WORLD);
        result = tmp;
        free(B_chk);
    }

    CUDA_CALL_SAFE(hipFree(d_A));
    CUDA_CALL_SAFE(hipFree(d_B));

    if (FTI_APP_RANK == 0 && (state == RESTART || state == KEEP)) {
        if (result == 0) {
            printf("[SUCCESSFUL]\n");
        } else {
            printf("[NOT SUCCESSFUL]\n");
            success=0;
        }
    }

    MPI_Barrier(FTI_COMM_WORLD);
    FTI_Finalize();
    MPI_Finalize();

    if (success == 1)
        return 0;
    else
        exit(DATA_CORRUPT);

}

/**
 * function definitions
 */

void init_arrays(double* A, double* B, size_t asize) {
    int i;
    for (i = 0; i< asize; i++) {
        A[i] = 1.0;
        B[i] = ((double)rand()/RAND_MAX)*5.0;
    }
}

__global__
void vecmult(double* A, double* B, size_t asize) {
    size_t i = (size_t)blockIdx.x * (size_t)blockDim.x + (size_t)threadIdx.x;
    if (i < asize) {
        A[i] = A[i] * B[i];
    }
}

int validify(double* A, double* B_chk, size_t asize) {
    int i;
    for (i=0; i<asize; i++) {
        if (A[i] != B_chk[i]){
            return -1;
        }
    }
    return 0;
}

int write_data(double* B, size_t *asize, int rank) {
    char str[256];
    sprintf(str, "chk/check-%i.tst", rank);
    FILE* f = fopen(str, "wb");
    size_t written = 0;

    fwrite( (void*) asize, sizeof(size_t), 1, f);

    while ( written < (*asize) ) {
        written += fwrite( (void*) B, sizeof(double), (*asize), f);
    }

    fclose(f);

    return 0;
}

int read_data(double* B_chk, size_t *asize_chk, int rank, size_t asize) {
    char str[256];
    sprintf(str, "chk/check-%i.tst", rank);
    FILE* f = fopen(str, "rb");
    size_t read = 0;

    fread( (void*) asize_chk, sizeof(size_t), 1, f);
    if ((*asize_chk) != asize) {
        printf("[ERROR -%i] : wrong dimension 'asize' -- asize: %zd, asize_chk: %zd\n", rank, asize, *asize_chk);
        fflush(stdout);
        return -1;
    }
    while ( read < (*asize_chk) ) {
        read += fread( (void*) B_chk, sizeof(double), (*asize_chk), f);
    }

    fclose(f);

    return 0;
}
